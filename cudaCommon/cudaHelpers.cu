#include "hip/hip_runtime.h"
#include <numeric>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "cudaHelpers.hpp"

enum class HyperplaneType {
    sP = 0,
    sN = 1,
    sL = 2,
}

__global__ void computeHyperplanes1(double* cyInd, double* scriptyH, int* sP, int* sN, 
                const int sPLen, const int sNLen, const int d, double* newHyp);

__global__ void partitionHyperplanes(double *C, HyperplaneType *hType, const double tol, 
                const int N);

// TODO Check CUDA Status
void cuFourierMotzkin(cudaHandles handles, double* x, double** scriptyH, int* scriptyHLen,
                double** workspace, const int workspaceLen, const int yInd, 
                const int n, const int d) {
    const double TOLERANCE = sqrt(std::numeric_limits<double>::epsilon());
    const int numHyperplanes = (*scriptyH)/d;
    const int iLenPart = 1024;
    int *sN;
    int *sP;
    int *sL;
    double *C;
    HyperplaneType *hType;

    // Allocate Data
    hipMalloc((void **)&C, sizeof(double)*(yInd + 1)*numHyperplanes);
    hipMalloc((void **)&hType, sizeof(HyperplaneType)*numHyperplanes);

    gpuMatmul(handles.ltHandle, x, scriptyH, C, yInd + 1, numHyperplanes, d, 
                    true, false, nullptr, 0);

    // Setup grid and block dimensions for partitioning
    dim3 block(iLen);
    dim3 grid(((*scriptyHLen/d)+block.x-1)/block.x);
    partitionHyperplanes<<<grid, block>>>(C, hType, TOLERANCE, yInd, numHyperplanes);
    
    
}

// cyInd = C at yInd
__global__ void computeHyperplanes1(double* C, double* scriptyH, int* sP, int* sN, 
                const int sPLen, const int sNLen, const int yInd, const int d, double* newHyp) {
    // TODO Try other possible combinations
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k;

    double lambda_i = C[(yInd + 1)*sP[i] + yInd];
    double lambda_j = C[(yInd + 1)*sN[j] + yInd]; // TODO Check this isn't a bug

    int tmpLen = i*d*sNLen;

    for (k = 0; k < d; k++) {
        newHyp[tmpLen + j*d + k] = lambda_i * scriptyH[sN[j]*d + k] - 
                                    lambda_j * scriptyH[sP[i]*d + k];
    }
}

__global__ void partitionHyperplanes(double *C, HyperplaneType *hType, const double tol, 
                const int yInd, const int N) {
    const int m = yInd + 1;
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = idx; i < N; i += gridDim.x*blockDim.x) {
        if (abs(C[i*m + yInd]) < tol) {
            hType[i] = HyperplaneType::sL;
        } else if(C[i*m + yInd] > tol) {
            hType[i] = HyperplaneType::sP;
        } else {
            hType[i] = HyperplaneType::sN;
        }
    }
}

hipblasStatus_t gpuMatmul(hipblasLtHandle_t handle, const double* A, const double* B, double* C,
                 const int m, const int n, const int k, const bool ta, const bool tb,
                 void* workspace, const size_t workspaceSize) {
    const int lda = ta ? k : m;
    const int ldb = tb ? n : k;
    const int ldc = m;
    int returnedResults = 0;
    hipblasStatus_t status;
    hipblasOperation_t transA = ta ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transB = tb ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc=NULL;
    hipblasLtMatmulPreference_t preference = NULL;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    // create operation description
    status = hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_64F, HIP_R_64F);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatmulDescSetAttribute(operationDesc, 
                                    HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(transA));
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatmulDescSetAttribute(operationDesc, 
                                    HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(transB));
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }

    // Create matrix descriptors
    status = hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_64F, 
                            transA == HIPBLAS_OP_N ? m : k, transA == HIPBLAS_OP_N ? k : m, lda);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_64F, 
                            transB == HIPBLAS_OP_N ? k : n, transB == HIPBLAS_OP_N ? n : k, ldb);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_64F, m, n, ldc);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }

    // Create preference handle
    status = hipblasLtMatmulPreferenceCreate(&preference);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatmulPreferenceSetAttribute(preference, 
                                HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, 
                                sizeof(workspaceSize));
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }

    // get the best available heuristic to try and run matmul
    status = hipblasLtMatmulAlgoGetHeuristic(handle, operationDesc, Adesc, Bdesc,
                        Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }

    if (returnedResults == 0) {
        return HIPBLAS_STATUS_NOT_SUPPORTED;
    }

    // Run matmul
    double alpha = 1.0f;
    double beta = 0.0f;
    status = hipblasLtMatmul(handle,
                             operationDesc,
                             &alpha, // alpha
                             A,
                             Adesc,
                             B,
                             Bdesc,
                             &beta, // beta
                             C,
                             Cdesc,
                             C,
                             Cdesc,
                             &heuristicResult.algo,
                             workspace,
                             workspaceSize,
                             0);

    if (preference) hipblasLtMatmulPreferenceDestroy(preference);
    if (Cdesc) hipblasLtMatrixLayoutDestroy(Cdesc);
    if (Bdesc) hipblasLtMatrixLayoutDestroy(Bdesc);
    if (Adesc) hipblasLtMatrixLayoutDestroy(Adesc);
    if (operationDesc) hipblasLtMatmulDescDestroy(operationDesc);

    return status;
}
