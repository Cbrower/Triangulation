#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "cudaHelpers.hpp"

enum class HyperplaneType {
    sP = 0,
    sN = 1,
    sL = 2,
}

__global__ void computeHyperplanes1(double* cyInd, double* scriptyH, int* sP, int* sN, 
                const int sPLen, const int sNLen, const int d, double* newHyp);
__global__ void partitionHyperplanes();

void cuFourierMotzkin(cudaHandles handles, double* x, double* d_x, double* scriptyH, 
            int* scriptyHLen, double** workspace, int yInd, const int n, const int d) {
    double *C;
    short *hType; // TODO Make this a smaller datatype maybe enum class

    // Allocate Matrix C
    C = hipMalloc((void **)&C, sizeof(double)*(yInd + 1)*(scriptyHLen/d));
    type = hipMalloc((void **)&type

    gpuMatmul(handles.ltHandle, x, scriptyH, C, yInd + 1, (*scriptyHLen) / d, d, 
                    true, false, nullptr, 0);
    
    
}

// TODO Understand cyInd
__global__ void computeHyperplanes1(double* cyInd, double* scriptyH, int* sP, int* sN, 
                const int sPLen, const int sNLen, const int d, double* newHyp) {
    // TODO Try other possible combinations
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k;

    double lambda_i = cyInd[sP[i]]; // What is cyInd
    double lambda_j = cyInd[sN[i]]; // TODO Check this isn't a bug

    int tmpLen = i*d*sNLen;

    for (k = 0; k < d; k++) {
        newHyp[tmpLen + j*d + k] = lambda_i * scriptyH[sN[j]*d + k] - 
                                    lambda_j * scriptyH[sP[i]*d + k];
    }
}

__global__ void partitionHyperplanes() {

}

hipblasStatus_t gpuMatmul(hipblasLtHandle_t handle, const double* A, const double* B, double* C,
                 const int m, const int n, const int k, const bool ta, const bool tb,
                 void* workspace, const size_t workspaceSize) {
    const int lda = ta ? k : m;
    const int ldb = tb ? n : k;
    const int ldc = m;
    int returnedResults = 0;
    hipblasStatus_t status;
    hipblasOperation_t transA = ta ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transB = tb ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc=NULL;
    hipblasLtMatmulPreference_t preference = NULL;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    // create operation description
    status = hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_64F, HIP_R_64F);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatmulDescSetAttribute(operationDesc, 
                                    HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(transA));
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatmulDescSetAttribute(operationDesc, 
                                    HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(transB));
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }

    // Create matrix descriptors
    status = hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_64F, 
                            transA == HIPBLAS_OP_N ? m : k, transA == HIPBLAS_OP_N ? k : m, lda);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_64F, 
                            transB == HIPBLAS_OP_N ? k : n, transB == HIPBLAS_OP_N ? n : k, ldb);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_64F, m, n, ldc);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }

    // Create preference handle
    status = hipblasLtMatmulPreferenceCreate(&preference);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatmulPreferenceSetAttribute(preference, 
                                HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, 
                                sizeof(workspaceSize));
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }

    // get the best available heuristic to try and run matmul
    status = hipblasLtMatmulAlgoGetHeuristic(handle, operationDesc, Adesc, Bdesc,
                        Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }

    if (returnedResults == 0) {
        return HIPBLAS_STATUS_NOT_SUPPORTED;
    }

    // Run matmul
    double alpha = 1.0f;
    double beta = 0.0f;
    status = hipblasLtMatmul(handle,
                             operationDesc,
                             &alpha, // alpha
                             A,
                             Adesc,
                             B,
                             Bdesc,
                             &beta, // beta
                             C,
                             Cdesc,
                             C,
                             Cdesc,
                             &heuristicResult.algo,
                             workspace,
                             workspaceSize,
                             0);

    if (preference) hipblasLtMatmulPreferenceDestroy(preference);
    if (Cdesc) hipblasLtMatrixLayoutDestroy(Cdesc);
    if (Bdesc) hipblasLtMatrixLayoutDestroy(Bdesc);
    if (Adesc) hipblasLtMatrixLayoutDestroy(Adesc);
    if (operationDesc) hipblasLtMatmulDescDestroy(operationDesc);

    return status;
}
