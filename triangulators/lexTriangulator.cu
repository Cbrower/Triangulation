#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <cmath>
#include <exception>
#include <algorithm>
#include <iterator>
#include <numeric>
#include <hip/hip_runtime.h>

#include "lexTriangulator.hpp"
#include "common.hpp"

const double TOLERANCE = sqrt(std::numeric_limits<double>::epsilon());

extern "C" {
    // LU decomoposition of a general matrix
    void dgetrf_(int* m, int *n, double* A, int* lda, int* ipiv, int* info);

    // generate inverse of a matrix given its LU decomp
    void dgetri_(int* n, double* A, int* lda, int* ipiv, double* work, int* lwork, int* info);

    // matrix vector product.  Note that this uses is in column major order
    void dgemv_(char* trans, int* m, int *n, double* alpha, double* A, int* lda, double* x, int* incx, double* beta, double* y, int* incy);

    // matrix matrix product.  Once again, this is in column major order
    void dgemm_(char* transA, char* transB, int* m, int* n, int* k, double* alpha, double* A, int* lda, double* B, int* ldb, double* beta, double* C, int* ldc);

    // Singular Value Decomposition of a matrix.
    void dgesvd_(char* jobu, char* jobvt, int* m, int* n, double* A, int* lda, double* S, 
                    double* U, int* ldu, double* VT, int* ldvt, double* work, int* lwork,
                    int* info);
}

// Helper Functions
int gcd(int a, int b);

void LexTriangulator::computeTri() {
    int i;
    int j;
    // scriptyH computation variables
    int lwspace;
    int error;
    int *piv;
    double det;
    double *lpckWspace;
    unsigned int flags;

    // setup initial values
    lwspace = d*d;

    // Allocate memory
    lpckWspace = new double[lwspace];
    // -- Allocate mem handle memory
    lenA = 2*d*d;
    A = new double[lenA];
    lenC = n*n;
    C = new double[lenC];
    lenD = n*n;
    D = new double[lenD];
    lenHyp = n*n;
    newHyp = new double[lenHyp]; // Currently not being used
    lenS = d;
    S = new double[lenS];
    lenWork = 5*d;
    work = new double[lenWork];
    piv = new int[d];

    scriptyHCap = d*n; // Starting size of scriptyH
    scriptyHLen = 0;
    // Allocate Zero Copy Memory TODO use preprocessor commands to make CUDA optional
    // scriptyH = new double[scriptyHCap];
    flags = hipHostMallocMapped;
    hipHostAlloc((void **)&scriptyH, scriptyHCap*sizeof(double), flags);

    for (i = 0; i < d; i++) {
        delta.push_back(i);
        for (j = 0; j < d; j++) {
            scriptyH[j*d + i] = x[i*d + j];
        }
    
    }

    dgetrf_(&d, &d, scriptyH, &d, piv, &error);
    if (error != 0) {
        throw std::runtime_error("Error in dgetrf");
    }

    det = 1;
    for (i = 0; i < d; i++) {
        det *= scriptyH[i*d + i];
        if (piv[i] != i+1) {
            det *= -1;
        }
    }

    dgetri_(&d, scriptyH, &d, piv, lpckWspace, &lwspace, &error);
    if (error != 0) {
        throw std::runtime_error("Error in dgetri");
    }

    // Scaling the rows of scriptyH
    for (i = 0; i < d; i++) {
        for (j = 0; j < d; j++) {
            scriptyH[i*d + j] *= det;
        }
    }

    // increment scriptyHLen to account for new data
    scriptyHLen += d*d;

    for (i = d; i < n; i++) {
        // extendTri(i);
        findNewHyp(i);
    }

    lexSort(scriptyH, scriptyHLen/d, d);

    computedTri = true;

    // Free memory
    delete[] piv;
    delete[] lpckWspace;
    delete[] A;
    delete[] C;
    delete[] D;
    delete[] newHyp;
    delete[] S;
    delete[] work;

    // Set lengths to zero
    lenA = 0;
    lenC = 0;
    lenD = 0;
    lenHyp = 0;
    lenS = 0;
    lenWork = 0;
}

void LexTriangulator::extendTri(int yInd) {
    // common
    std::vector<int> indTracker;
    int m;
    double alpha;
    int lda;
    double beta;
    // for determining \mathcal{H}^<(y) via matrix vector product of scriptyH and y
    char trans;
    double *p;
    int incx;
    int incy;
    // for \sigma \cap H via matrix matrix product
    char transA;
    char transB;
    int n1;
    int k;
    int ldb;
    double *C;
    int ldc;

    // TODO remove unnecesary parameters and put things in terms of d, n, etc.
    // Setting values for the \mathcal{H}^<(y) computation
    trans = 'T';
    m = d;
    n1 = scriptyHLen/d;
    alpha = 1.0;
    lda = m;
    incx = 1;    
    beta = 0.0;
    p = new double[n];
    incy = 1;

    dgemv_(&trans, &m, &n1, &alpha, scriptyH, &lda, &(x[yInd*d]), &incx, &beta, p, &incy);

    // setting values for the computation of \sigma \cap H
    transA = 'T';
    transB = 'N';
    m = yInd+1;
    n1 = scriptyHLen/d;
    k = d;
    lda = k;
    ldb = k;
    C = new double[n1*m];
    ldc = m;

    dgemm_(&transA, &transB, &m, &n1, &k, &alpha, x, &lda, scriptyH, &ldb, &beta, C, &ldc);

    // Can be parallelized
    int oDeltaLen = delta.size()/d;
    indTracker.reserve(d);
    for (int ih = 0;  ih < scriptyHLen/d; ih++) {
        if (p[ih] > -TOLERANCE) {
            continue;
        }

        for (int id = 0; id < oDeltaLen; id++) {
            indTracker.clear();
            for (int is = 0; is < d; is++) {
                if (fabs(C[ih*m + delta[id*d + is]]) < TOLERANCE) {
                    indTracker.push_back(delta[id*d + is]);
                }
            }

            if ((int)indTracker.size() == d - 1) {
                // TODO Use STL Functions
                for (int i = 0; i < d - 1; i++) {
                    delta.push_back(indTracker[i]);
                }
                delta.push_back(yInd);
            }
        }
    }

    delete[] p;
    delete[] C;
}

void LexTriangulator::findNewHyp(int yInd) {
    std::vector<int> sP;
    std::vector<int> sN;
    std::vector<int> sL;
    int i;
    int j;
    int k;
    int scale;
    char transA;
    char transB;
    int m;
    int n1;
    int k1;
    int lda;
    int ldb;
    int ldc;
    double alpha;
    double beta;
    // For New Hyperplanes
    int cap;
    int len;
    double lambda_i;
    double lambda_j;
    double *newHyp;
    // For filtering new hyperplanes
    int count;
    std::vector<int> toRemove; 
    // SVD for filtering new hyperplanes
    char jobu;
    char jobvt;
    int m2;
    int n2;
    int min;
    double *tmpA;
    int rowsA; // The number of rows A can have
    double* U;
    int ldu;
    double* VT;
    int ldvt;
    int lwork;
    int info;
    int numSVals;
    // For CUDA
    unsigned int flags;

    // For dgemm
    transA = 'T';
    transB = 'N';
    m = yInd+1;
    n1 = scriptyHLen/d;
    k1 = d;
    lda = k1;
    ldb = k1;
    if (n1*m > lenC) {
        scale = (int)(n1*m/lenC) + 1;
        delete[] C;
        lenC *= scale;
        C = new double[lenC];
    }
    ldc = m;
    alpha = 1.0;
    beta = 0.0;

    // For SVD
    jobu  = 'N';
    jobvt = 'N';
    m2 = d;
    rowsA = lenA/d;
    ldu = m2;
    ldvt = m2;
    U = nullptr;
    VT = nullptr;

    // TODO do the MM product outside of this function and extendTri function and use it commonly
    // for both
    dgemm_(&transA, &transB, &m, &n1, &k1, &alpha, x, &lda, scriptyH, &ldb, &beta, C, &ldc);

    for (i = 0; i < n1; i++) {
        if (fabs(C[i*m + yInd]) < TOLERANCE) {
            sL.push_back(i);
        } else if(C[i*m + yInd] > TOLERANCE) {
            sP.push_back(i);
        } else {
            sN.push_back(i);
        }
    }

    // 1) Allocate enough memory for our new forms
    cap = (sP.size() + sL.size() + sP.size()*sN.size())*d;
    len = 0;

    // Allocate Zero Copy Memory TODO use preprocessor commands to make CUDA optional
    // newHyp = new double[cap];
    flags = hipHostMallocMapped;
    hipHostAlloc((void **)&newHyp, cap*sizeof(double), flags);
    if (newHyp == nullptr) {
        throw std::runtime_error("Unable to allocate space for new hyperplanes");
    }

    // 2) Place the set builder notation elements from Theorem 7 of arxiv.0910.2845
    // into the newHyp array TODO Parallelize
    for (i = 0; i < (int)sP.size(); i++) {
        lambda_i = C[sP[i]*m + yInd];
        for (j = 0; j < (int)sN.size(); j++) {
            lambda_j = C[sN[j]*m + yInd];
            for (k = 0; k < d; k++) {
                newHyp[len + k] = lambda_i * scriptyH[sN[j]*d + k] -
                                        lambda_j * scriptyH[sP[i]*d + k];
            }
            len += d;
        }
    }

    // Remove Hyperplanes that do not have at least d-1 elements of x touching them
    // First, do a matrix matrix product of newHyp and 
    n1 = len/d;
    if (n1*m > lenD) {
        scale = (int)(n1*m/lenD) + 1; // TODO analyze and see if we want more than +1
        delete[] D;
        lenD *= scale;
        D = new double[lenD];
    }
    D = D;
    dgemm_(&transA, &transB, &m, &n1, &k1, &alpha, x, &lda, newHyp, &ldb, &beta, D, &ldc);

    // SVD Params
    A = A; // new double[2*d*d];
    lda = m2;
    S = S;
    work = work; // new double[5*d];

    // Find rows that do not have at least d-1 zeros or do not contain points that live in 
    // d-1 dimensional space.  The first is easy to check, the latter is done via svd
    // TODO Determine if I should just always copy or copy afterwards if and only if we
    // have at least d-1 points
    for (i = 0; i < n1; i++) {
        count = 0;
        for (j = 0; j < m; j++) {
            if (fabs(D[i*m + j]) < TOLERANCE) {
                if (count >= rowsA) {
                    // Increase the size of A and copy data over
                    tmpA = new double[2*rowsA*d];
                    
                    std::copy(A, A+rowsA*d, tmpA);
                    rowsA *= 2;

                    delete[] A;
                    A = tmpA;
                    A = tmpA;
                    tmpA = nullptr;
                }
                std::copy(x+j*d, x+(j+1)*d, A+count*d); 
                count += 1;
            }
        }
        if (count < d-1) {
            toRemove.push_back(i);
            continue;
        }

        // Now we compute the SVD and the number of singular values to get the rank
        n2 = count;
        min = fmin(m2, n2);
        numSVals = 0;
        lwork = 5*min;

        dgesvd_(&jobu, &jobvt, &m2, &n2, A, &lda, S, U, &ldu, VT, &ldvt, work, &lwork, &info);

        numSVals = 0;
        for (k = 0; k < min; k++) {
            if (fabs(S[k]) > TOLERANCE) {
                numSVals += 1;
            }
        }

        if (numSVals < d-1) {
            toRemove.push_back(i);
            continue;
        }
    }

    // Remove the rows found in above
    // TODO Speed up the shifting
    for (i = 0; i < (int) toRemove.size(); i++) {
        for (j = (toRemove[i]-i)*d; j < len - d; j++) {
            newHyp[j] = newHyp[j + d];
        }
        len -= d;
    }


    // 3) Add in the sP and sL
    for (i = 0; i < (int)sP.size(); i++) {
        for (int j = 0; j < d; j++) {
            newHyp[len + j] = scriptyH[sP[i]*d + j];
        }
        len += d;
    }

    for (i = 0; i < (int)sL.size(); i++) {
        for (int j = 0; j < d; j++) {
            newHyp[len + j] = scriptyH[sL[i]*d + j];
        }
        len += d;
    }

    // 4) Reduce them by gcd TODO parallelize with omp
    for (i = 0; i < len/d; i++) {
        int listGCD = 0;
        for (j = 0; j < d; j++) {
            // First, round our double array b/c of fp errors.  This should always contain ints.
            newHyp[i*d + j] = round(newHyp[i*d + j]);
            listGCD = gcd(listGCD, abs(newHyp[i*d + j]));
        }

        for (j = 0; j < d; j++) {
            newHyp[i*d + j] /= listGCD;
        }
    }

    // Free old scriptyH and replace
    hipHostFree(scriptyH);
    scriptyH = newHyp;
    scriptyHLen = len;
    scriptyHCap = cap;
}

int gcd(int a, int b) {
    int tmp;
    while (b != 0) {
        tmp = b;
        b = a % b;
        a = tmp;
    }
    return a;
}
