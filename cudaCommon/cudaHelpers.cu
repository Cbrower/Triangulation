#include "hip/hip_runtime.h"
#include <numeric>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/sequence.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include "cudaHelpers.hpp"

const bool TRANS = true;

enum class HyperplaneType {
    sP = 0,
    sN = 1,
    sL = 2,
};

__global__ void computeHyperplanes1(double* C, double* scriptyH, int* sP, int* sN, 
                const int sPLen, const int sNLen, const int yInd, const int d, double* newHyp);

__global__ void partitionHyperplanes(double *C, HyperplaneType *hType, const double tol, 
                const int yInd, const int N);

__global__ void countNumIntersects(double* D, int* numPts, bool* mask, const int N, 
                    const int nPts, const int d, const double tol);

__global__ void loadHypData(double *workspace, const double* x, const double* D,
                int* fmHyps, const int offset, const int workspaceLen, const int maxNPts, 
                const int N, const int nPts, const int d, const double tol, 
                const bool trans=false);

__global__ void checkSingularVals(const int* info, const double* S, bool* mask, 
                const int batchSize, const int offset, const int minMN,
                const int d, const double tol);

__global__ void mappedCopyHyperplanes(double *output, const double *input, const int N, const int d, int* map);

__global__ void mappedCopyAndReduceHyps(double *output, const double *input, const int N, const int d, int* map);

template <typename T, typename S>
void gpuSortVecs(T* vec, S* keys, const int N) {
    thrust::device_ptr<T> t_vec(vec);
    thrust::device_ptr<S> t_keys(keys);
    thrust::stable_sort_by_key(t_keys, t_keys + N, t_vec);
}

template <typename T>
int gpuFindFirst(T* vec, T val, const int N) {
    int ind;
    thrust::device_ptr<T> t_vec(vec);
    ind = thrust::find(thrust::device, t_vec, t_vec + N, val) - t_vec;
    return (ind == N) ? -1 : ind;
}

template <typename T>
T gpuMax(T* vec, const int N) {
    thrust::device_ptr<T> t_vec(vec);
    return *thrust::max_element(t_vec, t_vec + N);
}

inline void checkCudaStatus(hipError_t status, int line) {
    if (status != hipSuccess) {
        printf("cuda API failed with status %d: %s on line %d in file: cudaHelpers.cu\n", 
                status, hipGetErrorString(status), line);
        throw std::logic_error("cuda API failed");
    }
}

inline void checkCusolverStatus(hipsolverStatus_t status) {
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        printf("cuSolver API failed with status %d\n", status);
        throw std::logic_error("cuSolver API failed");
    }
}

// TODO Check CUDA Status
void cuFourierMotzkin(cudaHandles handles, double* x, double** scriptyH, int* scriptyHLen,
                int* scriptyHCap, double* workspace, const int workspaceLen, const int yInd, 
                const int n, const int d) {
    const double TOLERANCE = sqrt(std::numeric_limits<float>::epsilon());
    const int numHyperplanes = (*scriptyHLen)/d;
    const int iLenPart = 1024;
    const int iLenFM = 32;
    int *numPts;
    int *hyps;
    int *sN;
    int *sP;
    int *sL;
    int *fmHyps;
    int maxNPts;
    int sNLen;
    int sPLen;
    int sLLen;
    int fmHypsLen;
    bool* bitMask;
    double *newHyps;
    double *C;
    double *D;
    dim3 block;
    dim3 grid;
    HyperplaneType *hType;

#if VERBOSE == 1
    std::cout << "cuFourierMotzkin with yInd = " << yInd << "\n";
#endif

    // Allocate Data
    checkCudaStatus(hipMalloc((void **)&C, 
                sizeof(double)*(yInd + 1)*numHyperplanes), __LINE__);
    checkCudaStatus(hipMalloc((void **)&hType, 
                sizeof(HyperplaneType)*numHyperplanes), __LINE__);
    checkCudaStatus(hipMalloc((void **)&hyps,
                sizeof(int)*numHyperplanes), __LINE__);

    // Initialize hyps as a sequence to associate hyperplanes and their HType
    // Similar to STL's iota
    thrust::sequence(thrust::device, hyps, hyps + numHyperplanes, 0);

    // Conduct the matrix multiplication
    gpuMatmul(handles.ltHandle, x, *scriptyH, C, yInd + 1, numHyperplanes, d, 
                    true, false, nullptr, 0);
    /*
    gpuMatmul(handles.ltHandle, x, *scriptyH, C, yInd + 1, numHyperplanes, d, 
                    true, false, workspace, workspaceLen*sizeof(double));
    */

    // Setup grid and block dimensions for partitioning
    block.x = iLenPart;
    grid.x = ((*scriptyHLen/d)+block.x-1)/block.x;
    // Partition the hyperplenes
    partitionHyperplanes<<<grid, block>>>(C, hType, TOLERANCE, yInd, numHyperplanes);
    checkCudaStatus(hipGetLastError(), __LINE__);
    checkCudaStatus(hipDeviceSynchronize(), __LINE__);

    // Sort hyps so that we group sP's together, sN's together, and sL's together
    gpuSortVecs(hyps, hType, numHyperplanes);
    // Compute the length of each vector
    sPLen = gpuFindFirst(hType, HyperplaneType::sN, numHyperplanes);
    sNLen = gpuFindFirst(hType, HyperplaneType::sL, numHyperplanes);
    if (sNLen == -1) {
        sNLen = numHyperplanes;
    }
    sNLen -= sPLen;
    sLLen = numHyperplanes - sNLen - sPLen;

    assert(sPLen + sLLen + sNLen == numHyperplanes);

    // Free now unneeded memory
    checkCudaStatus(hipFree(hType), __LINE__);

    // Point to the proper spots in memory for sP, sN, and sL
    sP = hyps;
    sN = hyps + sPLen;
    sL = sN + sNLen;

    // Generate the new block and grid dimensions
    // The x dimension is for sP and the y is for sN
    block.x = iLenFM;
    block.y = iLenFM;
    grid.x = max((sPLen+block.x-1)/block.x, 1);
    grid.y = max((sNLen+block.x-1)/block.x, 1);

    checkCudaStatus(hipMalloc((void **)&newHyps, 
                sizeof(double)*max(1, sPLen)*max(1, sNLen)*d), __LINE__);

    // Execute the kernel
    computeHyperplanes1<<<grid, block>>>(C, *scriptyH, sP, sN, sPLen, sNLen, 
                                            yInd, d, newHyps);
    checkCudaStatus(hipGetLastError(), __LINE__);
    checkCudaStatus(hipDeviceSynchronize(), __LINE__);

    // Free no longer needed memory
    checkCudaStatus(hipFree(C), __LINE__);

    // Run matrix multiplication of the newHyperplanes
    checkCudaStatus(
            hipMalloc((void **)&D, sizeof(double)*(yInd + 1)*(max(1, sPLen)*max(1, sNLen))), 
            __LINE__);
    checkCublasStatus(gpuMatmul(handles.ltHandle, x, newHyps, D, yInd + 1, sPLen*sNLen, d, 
                    true, false, nullptr, 0));

    // Get largest number of intersected points by one of the hyperplanes
    checkCudaStatus(hipMalloc((void **)&numPts, 
                sizeof(int)*sPLen*sNLen), __LINE__);
    checkCudaStatus(hipMalloc((void**)&bitMask, 
                sizeof(bool)*sPLen*sNLen), __LINE__);
    block.x = iLenPart; // TODO maybe change
    block.y = 1;
    grid.x = (sPLen*sNLen + block.x - 1)/block.x;
    grid.y = 1;
    countNumIntersects<<<grid, block>>>(D, numPts, bitMask, sPLen*sNLen, yInd+1, d, TOLERANCE);
    checkCudaStatus(hipGetLastError(), __LINE__);
    maxNPts = gpuMax(numPts, sPLen*sNLen);
    if (maxNPts <= d) {
        maxNPts = d+1;
    }

    // Partition elements that have at least d-1 points they touch from the rest.
    // TODO Check that this op is faster than skipping this step.
    checkCudaStatus(hipMalloc((void**)&fmHyps,
                sizeof(int)*sPLen*sNLen), __LINE__);
    thrust::sequence(thrust::device, fmHyps, fmHyps + sPLen*sNLen, 0);
    gpuSortVecs(fmHyps, bitMask, sPLen*sNLen);
    fmHypsLen = gpuFindFirst(bitMask, true, sPLen*sNLen);
    if (fmHypsLen == -1) {
        fmHypsLen = sPLen*sNLen;
    }

    // Run batchedSVD's to determine which are valid
    // NOTE: Each matrix is d by maxNPts
    int batchesLeft = fmHypsLen;
    int numBatchesPerIt = workspaceLen / (d*maxNPts);
    int initNumBatches = min(fmHypsLen, numBatchesPerIt);
    int offset = 0;
    int batchSize;

    // Allocate memory for the singular values
    int minMN = (d < maxNPts) ? d : maxNPts;
    double *S;
    int *info; 
    double *U;
    double *V;
    checkCudaStatus(hipMalloc((void **)&S, 
                sizeof(double)*initNumBatches*minMN), __LINE__);
    checkCudaStatus(hipMalloc((void **)&info,
                sizeof(int)*initNumBatches), __LINE__);
    if (TRANS) {
        checkCudaStatus(hipMalloc((void**)&U,
                    sizeof(double)*initNumBatches*maxNPts*maxNPts), __LINE__);
        checkCudaStatus(hipMalloc((void**)&V,
                    sizeof(double)*initNumBatches*d*d), __LINE__);
    } else {
        checkCudaStatus(hipMalloc((void**)&U,
                    sizeof(double)*initNumBatches*d*d), __LINE__);
        checkCudaStatus(hipMalloc((void**)&V,
                    sizeof(double)*initNumBatches*maxNPts*maxNPts), __LINE__);
    }

#if VERBOSE == 1
    std::cout << "MaxNPts = " << maxNPts << "\n";
    std::cout << "sPLen = " << sPLen << "\n";
    std::cout << "sNLen = " << sNLen << "\n";
    std::cout << "sLLen = " << sLLen << "\n";
    std::cout << "fmHypsLen = " << fmHypsLen << "\n";
    std::cout << "minMN = " << minMN << "\n";
#endif
    while (batchesLeft > 0) {
        batchSize = min(batchesLeft, numBatchesPerIt);
        // CUDA Kernel To Prepare Workspace
        block.x = iLenFM; // TODO Maybe change
        block.y = 1;
        grid.x = (batchSize + block.x - 1)/block.x;
        grid.y = 1;
        // TODO Ensure batchSize is the correct argument
        loadHypData<<<grid, block, maxNPts*block.x*sizeof(int)>>>(workspace, x, D, fmHyps, 
                offset, workspaceLen, maxNPts, batchSize, yInd+1, d, TOLERANCE, TRANS);
        checkCudaStatus(hipGetLastError(), __LINE__);
        checkCudaStatus(hipDeviceSynchronize(), __LINE__);

#if VERBOSE == 1
        {
            double* buf = new double[batchSize*maxNPts*d];
            hipMemcpy(buf, workspace, sizeof(double)*batchSize*maxNPts*d, 
                    hipMemcpyDeviceToHost);
            std::cout << "Workspace:\n";
            if (TRANS) {
                for (int i = 0; i < batchSize; i++) {
                    std::cout << "Matrix " << i << "\n";
                    for (int j = 0; j < maxNPts; j++) {
                        for (int k = 0; k < d; k++) {
                            std::cout << buf[i*maxNPts*d + k*maxNPts + j] << " ";
                        }
                        std::cout << "\n";
                    }
                }
            } else {
                for (int i = 0; i < batchSize; i++) {
                    std::cout << "Matrix " << i << "\n";
                    for (int j = 0; j < maxNPts; j++) {
                        for (int k = 0; k < d; k++) {
                            std::cout << buf[i*maxNPts*d + j*d + k] << " ";
                        }
                        std::cout << "\n";
                    }
                }
            }
            delete[] buf;
        }
#endif

        // Call cuSolver to get svd
        checkCusolverStatus(gpuBatchedGetApproxSingularVals(handles.dnHandle, workspace, S, info, maxNPts, d, batchSize, U, V));
        /*
        if (TRANS) {
            checkCusolverStatus(gpuBatchedGetSingularVals(handles.dnHandle, workspace, S, info, maxNPts, d, batchSize, U, V));
        } else {
            checkCusolverStatus(gpuBatchedGetSingularVals(handles.dnHandle, workspace, S, info, d, maxNPts, batchSize, U, V));
        }
        */

#if VERBOSE == 1
        {
            int *buf = new int[batchSize];
            hipMemcpy(buf, info, sizeof(int)*batchSize, hipMemcpyDeviceToHost);
            std::cout << "Info:\n";
            for (int i = 0; i < batchSize; i++) {
                std::cout << buf[i] << " ";
            }
            std::cout << "\n";
            delete[] buf;
        }
#endif

#if VERBOSE == 1
        {
            double *buf = new double[batchSize*d];
            hipMemcpy(buf, S, sizeof(double)*batchSize*d, hipMemcpyDeviceToHost);
            std::cout << "singular vals:\n";
            for (int i = 0; i < batchSize; i++) {
                for (int j = 0; j < d; j++) {
                    std::cout << buf[i*d + j] << " ";
                }
                std::cout << "\n";
            }
            std::cout << "\n";
            delete[] buf;
        }
#endif

#if VERBOSE == 1
        {
            bool *buf = new bool[sPLen*sNLen];
            hipMemcpy(buf, bitMask, sizeof(bool)*sPLen*sNLen, hipMemcpyDeviceToHost);
            std::cout << "bitMask before:\n";
            for (int i = 0; i < sPLen*sNLen; i++) {
                std::cout << buf[i] << " ";
            }
            std::cout << "\n";
            delete[] buf;
        }
#endif

        // CUDA Kernel to update bitmask
        // NOTE prior block and grid dimensions work for this kernel
        checkSingularVals<<<grid, block>>>(info, S, bitMask, batchSize, offset, minMN, d, TOLERANCE);
        checkCudaStatus(hipGetLastError(), __LINE__);
        checkCudaStatus(hipDeviceSynchronize(), __LINE__);
#if VERBOSE == 1
        {
            bool *buf = new bool[sPLen*sNLen];
            hipMemcpy(buf, bitMask, sizeof(bool)*sPLen*sNLen, hipMemcpyDeviceToHost);
            std::cout << "bitMask after:\n";
            for (int i = 0; i < sPLen*sNLen; i++) {
                std::cout << buf[i] << " ";
            }
            std::cout << "\n";
            delete[] buf;
        }
#endif

        batchesLeft -= batchSize;
        offset += batchSize;
    }

    // Sort based on bitMask
    gpuSortVecs(fmHyps, bitMask, sPLen*sNLen);
    fmHypsLen = gpuFindFirst(bitMask, true, sPLen*sNLen);
    if (fmHypsLen == -1) {
        fmHypsLen = sPLen*sNLen;
    }

    double *nScriptyH;
    checkCudaStatus(hipMalloc((void **)&nScriptyH,
                sizeof(double)*d*(fmHypsLen + sPLen + sLLen)), __LINE__);

    // Copy sP
    block.x = iLenFM; // TODO Maybe change
    block.y = 1;
    grid.x = (sPLen + block.x - 1)/block.x;
    grid.y = 1;
    mappedCopyHyperplanes<<<grid, block>>>(nScriptyH, *scriptyH, sPLen, d, sP);
    checkCudaStatus(hipGetLastError(), __LINE__);
    checkCudaStatus(hipDeviceSynchronize(), __LINE__);

    // Copy sL
    if (sLLen > 0) {
        block.x = iLenFM; // TODO Maybe change
        grid.x = (sLLen + block.x - 1)/block.x;
        mappedCopyHyperplanes<<<grid, block>>>(nScriptyH + sPLen*d, *scriptyH, sLLen, d, sL);
        checkCudaStatus(hipGetLastError(), __LINE__);
        checkCudaStatus(hipDeviceSynchronize(), __LINE__);
    }

    // Copy FM Hyperplanes
    if (fmHypsLen > 0) {
        block.x = iLenFM; // TODO Maybe change
        grid.x = (fmHypsLen + block.x - 1)/block.x;
        mappedCopyAndReduceHyps<<<grid, block>>>(nScriptyH + (sPLen + sLLen)*d, 
                newHyps, fmHypsLen, d, fmHyps);
        checkCudaStatus(hipGetLastError(), __LINE__);
        checkCudaStatus(hipDeviceSynchronize(), __LINE__);
    }

    // Update scriptyH
    hipFree(*scriptyH);
    *scriptyH = nScriptyH;
    *scriptyHLen = d*(fmHypsLen + sPLen + sLLen);
    *scriptyHCap = *scriptyHLen;

#if VERBOSE == 1
    {
        double *buf = new double[*scriptyHLen];
        int numRows = (*scriptyHLen) / d;
        hipMemcpy(buf, *scriptyH, sizeof(double)*(*scriptyHLen), hipMemcpyDeviceToHost);
        
        std::cout << "scriptyH:\n";
        for (int i = 0; i < numRows; i++) {
            for (int j = 0; j < d; j++) {
                std::cout << buf[i*d + j] << " ";
            }
            std::cout << "\n";
        }
        delete[] buf;
    }
#endif

    // Free memory
    hipFree(D);
    hipFree(bitMask);
    hipFree(fmHyps);
    hipFree(hyps);
    hipFree(numPts);
}

// cyInd = C at yInd
__global__ void computeHyperplanes1(double* C, double* scriptyH, int* sP, int* sN, 
                const int sPLen, const int sNLen, const int yInd, const int d, double* newHyp) {
    // TODO Try other possible combinations
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k;
    double lambda_i;
    double lambda_j;

    if (i < sPLen && j < sNLen) {
        lambda_i = C[(yInd + 1)*sP[i] + yInd];
        lambda_j = C[(yInd + 1)*sN[j] + yInd]; // TODO Check this isn't a bug

        int tmpLen = i*d*sNLen;

        for (k = 0; k < d; k++) {
            newHyp[tmpLen + j*d + k] = lambda_i * scriptyH[sN[j]*d + k] - 
                                        lambda_j * scriptyH[sP[i]*d + k];
        }
    }
}

__global__ void partitionHyperplanes(double *C, HyperplaneType *hType, const double tol, 
                const int yInd, const int N) {
    const int m = yInd + 1;
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = idx; i < N; i += gridDim.x*blockDim.x) {
        // TODO Update to avoid warp divergence
        if (abs(C[i*m + yInd]) < tol) {
            hType[i] = HyperplaneType::sL;
        } else if(C[i*m + yInd] > tol) {
            hType[i] = HyperplaneType::sP;
        } else {
            hType[i] = HyperplaneType::sN;
        }
    }
}

__global__ void countNumIntersects(double* D, int* numPts, bool* mask, const int N, 
                    const int nPts, const int d, const double tol) {
    int i;
    int j;
    int count;
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (i = tid; i < N; i += gridDim.x*blockDim.x) {
        count = 0;
        for (j = 0; j < nPts; j++) {
            count += (int)(abs(D[i*nPts + j]) < tol);
        }
        numPts[i] = count;
        mask[i] = count < d - 1;
    }
}

// Assumes inds is a numThreadsPerBlock by maxNPts array
__global__ void loadHypData(double *workspace, const double* x, const double* D,
                int* fmHyps, const int offset, const int workspaceLen, const int maxNPts, 
                const int N, const int nPts, const int d, const double tol, const bool trans) {
    extern __shared__ int inds[];
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    int j;
    int val;
    int counter;
    int stride;

    if (idx >= N) {
        return;
    }

    counter = 0;
    stride = min(N, blockDim.x);

    // Part 1 Place the indices into the inds array
    for (i = 0; i < nPts; i++) {
        val = (int)(abs(D[fmHyps[idx + offset]*nPts + i]) < tol);
        inds[counter*stride + threadIdx.x] = i*val; // Could also try "idx*stride + counter"
        counter += val;
    }

    // Part 2 update indices to set to a valid value if i >= counter TODO Check if needed
    for (i = 0; i < maxNPts; i++) {
        inds[i*stride + threadIdx.x] = (int)(i < counter) * inds[i*stride + threadIdx.x];
    }

    // Part 3 Do the copy
    if (trans) {
        for (i = 0; i < d; i++) {
            for (j = 0; j < maxNPts; j++)  {
                workspace[idx*maxNPts*d + i*maxNPts + j] = (int)(j < counter) * 
                    x[inds[j*stride + threadIdx.x]*d + i];
            }
        }
    } else {
        for (i = 0; i < maxNPts; i++)  {
            for (j = 0; j < d; j++) {
                workspace[(idx*maxNPts + i)*d + j] = (int)(i < counter) * 
                    x[inds[i*stride + threadIdx.x]*d + j];
            }
        }
    }
}

__global__ void checkSingularVals(const int* info, const double* S, bool* mask, 
                const int batchSize, const int offset, const int minMN,
                const int d, const double tol) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int cnt;

    if (tid >= batchSize) {
        return;
    }

    for (int i = 0; i < minMN; i++) {
        cnt += (int)(abs(S[tid*minMN + i]) >= tol);
    }
    
    mask[tid + offset] = cnt < (d - 1);
}

__global__ void mappedCopyHyperplanes(double *output, const double *input, const int N, const int d, int* map) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int idx;

    if (tid < N) {
        idx = map[tid];
        for (int i = 0; i < d; i++) {
            output[tid*d + i] = input[idx*d + i];
        }
    }
}

// TODO Speedup with faster gcd algorithm
__device__ int gpuGCD(int a, int b) {
    int tmp;
    while (b != 0) {
        tmp = b;
        b = a % b;
        a = tmp;
    }

    return a;
}

__global__ void mappedCopyAndReduceHyps(double *output, const double *input, const int N, const int d, int* map) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int idx;
    int listGCD = 0;

    if (tid < N) {
        idx = map[tid];
        for (int i = 0; i < d; i++) {
            listGCD = gpuGCD(listGCD, abs(round(input[idx*d + i])));
        }
        for (int i = 0; i < d; i++) {
            output[tid*d + i] = round(input[idx*d + i])/listGCD;
        }
    }
}

hipblasStatus_t gpuMatmul(hipblasLtHandle_t handle, const double* A, const double* B, double* C,
                 const int m, const int n, const int k, const bool ta, const bool tb,
                 void* workspace, const size_t workspaceSize) {
    const int lda = ta ? k : m;
    const int ldb = tb ? n : k;
    const int ldc = m;
    int returnedResults = 0;
    hipblasStatus_t status;
    hipblasOperation_t transA = ta ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transB = tb ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc=NULL;
    hipblasLtMatmulPreference_t preference = NULL;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    // create operation description
    status = hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_64F, HIP_R_64F);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatmulDescSetAttribute(operationDesc, 
                                    HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(transA));
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatmulDescSetAttribute(operationDesc, 
                                    HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(transB));
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }

    // Create matrix descriptors
    status = hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_64F, 
                            transA == HIPBLAS_OP_N ? m : k, transA == HIPBLAS_OP_N ? k : m, lda);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_64F, 
                            transB == HIPBLAS_OP_N ? k : n, transB == HIPBLAS_OP_N ? n : k, ldb);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_64F, m, n, ldc);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }

    // Create preference handle
    status = hipblasLtMatmulPreferenceCreate(&preference);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }
    status = hipblasLtMatmulPreferenceSetAttribute(preference, 
                                HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, 
                                sizeof(workspaceSize));
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }

    // get the best available heuristic to try and run matmul
    status = hipblasLtMatmulAlgoGetHeuristic(handle, operationDesc, Adesc, Bdesc,
                        Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        return status;
    }

    if (returnedResults == 0) {
        return HIPBLAS_STATUS_NOT_SUPPORTED;
    }

    // Run matmul
    double alpha = 1.0f;
    double beta = 0.0f;
    status = hipblasLtMatmul(handle,
                             operationDesc,
                             &alpha, // alpha
                             A,
                             Adesc,
                             B,
                             Bdesc,
                             &beta, // beta
                             C,
                             Cdesc,
                             C,
                             Cdesc,
                             &heuristicResult.algo,
                             workspace,
                             workspaceSize,
                             0);

    if (preference) hipblasLtMatmulPreferenceDestroy(preference);
    if (Cdesc) hipblasLtMatrixLayoutDestroy(Cdesc);
    if (Bdesc) hipblasLtMatrixLayoutDestroy(Bdesc);
    if (Adesc) hipblasLtMatrixLayoutDestroy(Adesc);
    if (operationDesc) hipblasLtMatmulDescDestroy(operationDesc);

    return status;
}

hipsolverStatus_t gpuBatchedGetSingularVals(hipsolverHandle_t cusolverH, double* A, double* S, int* info, 
        const int m, const int n, const int batchSize, double* U, double* V) {
    
    const bool nullU = U == nullptr;
    const bool nullV = U == nullptr;
    hipError_t cuStatus = hipSuccess;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipsolverGesvdjInfo_t gesvdj_params = NULL;
    double *d_work;
    const int lda = m; /* lda >= m */
    const int ldu = m; /* ldu >= m */
    const int ldv = n; /* ldv >= n */
    int lwork = 0;       /* size of workspace */

    const double tol = 1.e-7;
    const int max_sweeps = 15;
    const int sort_svd  = 0;   /* don't sort singular values */
    /* Don't compute singular vectors */
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;

    assert (1 <= m && m <= 32);
    assert (1 <= n && n <= 32);

    if (nullU) {
        cuStatus = hipMalloc((void**)&U, sizeof(double)*batchSize*ldu*m);
        if (cuStatus != hipSuccess) {
            return HIPSOLVER_STATUS_ALLOC_FAILED;
        }
    }
    if (nullV) {
        cuStatus = hipMalloc((void**)&V, sizeof(double)*batchSize*ldv*n);
        if (cuStatus != hipSuccess) {
            return HIPSOLVER_STATUS_ALLOC_FAILED;
        }
    }

/* step 2: configuration of gesvdj */
    status = hipsolverDnCreateGesvdjInfo(&gesvdj_params);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        return status;
    }

/* default value of tolerance is machine zero */
    status = hipsolverDnXgesvdjSetTolerance(
        gesvdj_params,
        tol);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        return status;
    }

/* default value of max. sweeps is 100 */
    status = hipsolverDnXgesvdjSetMaxSweeps(
        gesvdj_params,
        max_sweeps);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        return status;
    }

/* disable sorting */
    status = hipsolverDnXgesvdjSetSortEig(
        gesvdj_params,
        sort_svd);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        return status;
    }

/* step 4: query working space of gesvdjBatched */
    status = hipsolverDnDgesvdjBatched_bufferSize(
        cusolverH,
        jobz,
        m,
        n,
        A,
        lda,
        S,
        U,
        ldu,
        V,
        ldv,
        &lwork,
        gesvdj_params,
        batchSize
    );
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        return status;
    }

    cuStatus = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    if (cuStatus != hipSuccess) {
        return HIPSOLVER_STATUS_ALLOC_FAILED;
    }

/* step 5: compute singular values of A0 and A1 */
    status = hipsolverDnDgesvdjBatched(
        cusolverH,
        jobz,
        m,
        n,
        A,
        lda,
        S,
        U,
        ldu,
        V,
        ldv,
        d_work,
        lwork,
        info,
        gesvdj_params,
        batchSize
    );
    cuStatus = hipDeviceSynchronize();
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        return status;
    }
    if (cuStatus != hipSuccess) {
        return HIPSOLVER_STATUS_ALLOC_FAILED;
    }

    if (gesvdj_params) hipsolverDnDestroyGesvdjInfo(gesvdj_params);
    if (nullU && U) hipFree(U);
    if (nullV && V) hipFree(V);
    if (d_work ) hipFree(d_work);

    return HIPSOLVER_STATUS_SUCCESS;
}

// Influenced from the official cusolver library samples
hipsolverStatus_t gpuBatchedGetApproxSingularVals(hipsolverHandle_t cusolverH, double* A, 
        double* S, int* info, const int m, const int n, const int batchSize, double* U, 
        double* V) {

    const int lda = m;
    const int ldu = m;
    const int ldv = n;
    const long long int strideA = static_cast<long long int>(lda * n);
    const long long int strideS = n;
    const long long int strideU = static_cast<long long int>(ldu * n);
    const long long int strideV = static_cast<long long int>(ldv * n);
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
    int rank = n;
    int lwork = 0;
    double *work = nullptr;
    hipError_t cuStatus;
    hipsolverStatus_t status;

    if (m <= n) {
        return HIPSOLVER_STATUS_INVALID_VALUE;
    }

    status = hipsolverDnDgesvdaStridedBatched_bufferSize(
            cusolverH, jobz, rank, m, n, A, lda, strideA,
            S, strideS, U, ldu, strideU, V, ldv, strideV,
            &lwork, batchSize);

    if (status != HIPSOLVER_STATUS_SUCCESS) {
        return status;
    }

    cuStatus = hipMalloc(reinterpret_cast<void **>(&work), sizeof(double)*lwork);
    
    if (cuStatus != hipSuccess) {
        return HIPSOLVER_STATUS_ALLOC_FAILED;
    }

    status = hipsolverDnDgesvdaStridedBatched(
            cusolverH, jobz, rank, m, n, A, lda, strideA,
            S, strideS, U, ldu, strideU, V, ldv, strideV,
            work, lwork, info, nullptr, batchSize);

    hipFree(work);
    return status;
}
